/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 16

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     * Use shared memory for tiling
     *
     ********************************************************************/

    // INSERT KERNEL CODE HERE
    
    // Create two new tiles for Matrix A and Matrix B at shared memory
    __shared__ float aTile[TILE_SIZE][TILE_SIZE];
    __shared__ float bTile[TILE_SIZE][TILE_SIZE];
    
    // Initialize variables
    float temp = 0.0;
    int xBlock = blockIdx.x;
    int yBlock = blockIdx.y;
    int xThread = threadIdx.x;
    int yThread = threadIdx.y;
    int row = yBlock * TILE_SIZE + yThread;
    int col = xBlock * TILE_SIZE + xThread;
    
    // Perform calculations
    for(int i = 0; i < ((k + TILE_SIZE - 1) / TILE_SIZE); i++) {
    
        // A Tile
        if(xThread + (i * TILE_SIZE) < k && row < m) {
            aTile[yThread][xThread] = A[(row * k) + xThread + (i * TILE_SIZE)];
        }
        else {
            aTile[yThread][xThread] = 0.0f; // Set to zero
        }
    
        // B Tile
        if(yThread + (i * TILE_SIZE) < k && col < n) {
            bTile[yThread][xThread] = B[(i * TILE_SIZE + yThread) * n + col];
        }
        else {
            bTile[yThread][xThread] = 0.0f; // Set to zero
        }
        
        __syncthreads();    // Synchronization point; force threads to wait
        
        // Update temp value
        for(int z = 0; z < TILE_SIZE; z++) {
            temp += aTile[yThread][z] * bTile[z][xThread];
        }
        
        __syncthreads();    // Sychronization point; force threads to wait
    }
    
    // Copy temp value to C Matrix
    if(row < m && col < n) {
        C[row * n + col] = temp;
    }
}

void basicSgemm(char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc)
{
    if ((transa != 'N') && (transa != 'n')) {
	printf("unsupported value of 'transa'\n");
    	return;
    }

    if ((transb != 'N') && (transb != 'n')) {
	printf("unsupported value of 'transb'\n");
	return;
    }

    if ((alpha - 1.0f > 1e-10) || (alpha - 1.0f < -1e-10)) {
	printf("unsupported value of alpha\n");
	return;
    }

    if ((beta - 0.0f > 1e-10) || (beta - 0.0f < -1e-10)) {
	printf("unsupported value of beta\n");
	return;
    }

    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = TILE_SIZE;

    //INSERT CODE HERE
    dim3 dim_grid((n / BLOCK_SIZE) + 1, (m / BLOCK_SIZE) + 1);
    dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE);

    // Invoke CUDA kernel -----------------------------------------------------

    //INSERT CODE HERE
    mysgemm <<< dim_grid, dim_block >>> (m, n, k, A, B, C);
}